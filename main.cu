/*
 * keccak.cu  Implementation of Keccak/SHA3 digest
 *
 * Date: 12 June 2019
 * Revision: 1
 *
 * This file is released into the Public Domain.
 */
 
// Edited by krlnokrl
#include <hip/hip_runtime.h>

#include <hiprand/hiprand_kernel.h>
#include <pthread.h>
#include <stdint.h>
#include <stdio.h>
#include <unistd.h>
#include <cstdlib>
#include <ctime>
 
typedef unsigned char BYTE;
typedef unsigned int  WORD;
typedef unsigned long long LONG; 
 

#define KECCAK_ROUND 24
#define KECCAK_STATE_SIZE 25
#define KECCAK_Q_SIZE 192

#define N 2147483640

__constant__ LONG CUDA_KECCAK_CONSTS[24] = { 0x0000000000000001, 0x0000000000008082,
                                          0x800000000000808a, 0x8000000080008000, 0x000000000000808b, 0x0000000080000001, 0x8000000080008081,
                                          0x8000000000008009, 0x000000000000008a, 0x0000000000000088, 0x0000000080008009, 0x000000008000000a,
                                          0x000000008000808b, 0x800000000000008b, 0x8000000000008089, 0x8000000000008003, 0x8000000000008002,
                                          0x8000000000000080, 0x000000000000800a, 0x800000008000000a, 0x8000000080008081, 0x8000000000008080,
                                          0x0000000080000001, 0x8000000080008008 };

typedef struct {

    BYTE sha3_flag;
    WORD digestbitlen;
    LONG rate_bits;
    LONG rate_BYTEs;
    LONG absorb_round;

    int64_t state[KECCAK_STATE_SIZE];
    BYTE q[KECCAK_Q_SIZE];

    LONG bits_in_queue;

} cuda_keccak_ctx_t;
typedef cuda_keccak_ctx_t CUDA_KECCAK_CTX;


__device__ __forceinline__ LONG cuda_keccak_leuint64(void* const in)
{
    LONG a;
    memcpy(&a, in, 8);
    return a;
}


//#define cuda_keccak_MIN(a,b) ((a) < (b) ? (a) : (b))
//#define cuda_keccak_UMIN(a,b) ((a) < (b) ? (a) : (b))


__device__ __forceinline__ int64_t cuda_keccak_MIN(const int64_t a, const int64_t b)
{
    if (a > b) return b;
    return a;
}

__device__ __forceinline__ LONG cuda_keccak_UMIN(const LONG a, const LONG b)
{
    if (a > b) return b;
    return a;
}

__device__ __forceinline__ unsigned long long xor5(const unsigned long long a, const unsigned long long b, const unsigned long long c, const unsigned long long d, const unsigned long long e)
{
	unsigned long long result;
	asm("xor.b64 %0, %1, %2;" : "=l"(result) : "l"(d) ,"l"(e));
	asm("xor.b64 %0, %0, %1;" : "+l"(result) : "l"(c));
	asm("xor.b64 %0, %0, %1;" : "+l"(result) : "l"(b));
	asm("xor.b64 %0, %0, %1;" : "+l"(result) : "l"(a));
	return result;
}


__device__ void cuda_keccak_extract(cuda_keccak_ctx_t *ctx)
{
    LONG len = ctx->rate_bits >> 6;
    int64_t a;
    int s = sizeof(LONG);
	
	#pragma unroll 2
    for (int i = 0;i < len;i++) {
        a = cuda_keccak_leuint64((int64_t*)&ctx->state[i]);
        memcpy(ctx->q + (i * s), &a, s);
    }
}


__device__ __forceinline__ unsigned long long cuda_keccak_ROTL64(const unsigned long long x, const int offset) {
	unsigned long long res;
	asm("{ // ROTL64 \n\t"
		".reg .u32 tl,th,vl,vh;\n\t"
		".reg .pred p;\n\t"
		"mov.b64 {tl,th}, %1;\n\t"
		"shf.l.wrap.b32 vl, tl, th, %2;\n\t"
		"shf.l.wrap.b32 vh, th, tl, %2;\n\t"
		"setp.lt.u32 p, %2, 32;\n\t"
		"@!p mov.b64 %0, {vl,vh};\n\t"
		"@p  mov.b64 %0, {vh,vl};\n\t"
	"}\n" : "=l"(res) : "l"(x) , "r"(offset)
	);
	return res;
}
/*__device__ __forceinline__ LONG cuda_keccak_ROTL64(LONG a, LONG  b)
{
    return (a << b) | (a >> (64 - b));
}
*/



__device__ __forceinline__ static void cuda_keccak_permutations(cuda_keccak_ctx_t * ctx)
{

    int64_t* A = ctx->state;;

    int64_t *a00 = A, *a01 = A + 1, *a02 = A + 2, *a03 = A + 3, *a04 = A + 4;
    int64_t *a05 = A + 5, *a06 = A + 6, *a07 = A + 7, *a08 = A + 8, *a09 = A + 9;
    int64_t *a10 = A + 10, *a11 = A + 11, *a12 = A + 12, *a13 = A + 13, *a14 = A + 14;
    int64_t *a15 = A + 15, *a16 = A + 16, *a17 = A + 17, *a18 = A + 18, *a19 = A + 19;
    int64_t *a20 = A + 20, *a21 = A + 21, *a22 = A + 22, *a23 = A + 23, *a24 = A + 24;
	
	int64_t c0;
	int64_t c1;
	int64_t c2;
	int64_t c3;
	int64_t c4;
	
	int64_t d0;
	int64_t d1;
	int64_t d2;
	int64_t d3;
	int64_t d4;
	
	#pragma unroll 2
    for (int i = 0; i < KECCAK_ROUND; i++) {

        /* Theta */
        /*
		c0 = *a00 ^ *a05 ^ *a10 ^ *a15 ^ *a20;
        c1 = *a01 ^ *a06 ^ *a11 ^ *a16 ^ *a21;
        c2 = *a02 ^ *a07 ^ *a12 ^ *a17 ^ *a22;
        c3 = *a03 ^ *a08 ^ *a13 ^ *a18 ^ *a23;
        c4 = *a04 ^ *a09 ^ *a14 ^ *a19 ^ *a24;
		*/
		c0 = xor5(*a00, *a05, *a10, *a15, *a20);
		c1 = xor5(*a01, *a06, *a11, *a16, *a21);
		c2 = xor5(*a02, *a07, *a12, *a17, *a22);
		c3 = xor5(*a03, *a08, *a13, *a18, *a23);
		c4 = xor5(*a04, *a09, *a14, *a19, *a24);
		
        d1 = cuda_keccak_ROTL64(c1, 1) ^ c4;
        d2 = cuda_keccak_ROTL64(c2, 1) ^ c0;
        d3 = cuda_keccak_ROTL64(c3, 1) ^ c1;
        d4 = cuda_keccak_ROTL64(c4, 1) ^ c2;
        d0 = cuda_keccak_ROTL64(c0, 1) ^ c3;

        *a00 ^= d1;
        *a05 ^= d1;
        *a10 ^= d1;
        *a15 ^= d1;
        *a20 ^= d1;
        *a01 ^= d2;
        *a06 ^= d2;
        *a11 ^= d2;
        *a16 ^= d2;
        *a21 ^= d2;
        *a02 ^= d3;
        *a07 ^= d3;
        *a12 ^= d3;
        *a17 ^= d3;
        *a22 ^= d3;
        *a03 ^= d4;
        *a08 ^= d4;
        *a13 ^= d4;
        *a18 ^= d4;
        *a23 ^= d4;
        *a04 ^= d0;
        *a09 ^= d0;
        *a14 ^= d0;
        *a19 ^= d0;
        *a24 ^= d0;

        /* Rho pi */
        c1 = cuda_keccak_ROTL64(*a01, 1);
        *a01 = cuda_keccak_ROTL64(*a06, 44);
        *a06 = cuda_keccak_ROTL64(*a09, 20);
        *a09 = cuda_keccak_ROTL64(*a22, 61);
        *a22 = cuda_keccak_ROTL64(*a14, 39);
        *a14 = cuda_keccak_ROTL64(*a20, 18);
        *a20 = cuda_keccak_ROTL64(*a02, 62);
        *a02 = cuda_keccak_ROTL64(*a12, 43);
        *a12 = cuda_keccak_ROTL64(*a13, 25);
        *a13 = cuda_keccak_ROTL64(*a19, 8);
        *a19 = cuda_keccak_ROTL64(*a23, 56);
        *a23 = cuda_keccak_ROTL64(*a15, 41);
        *a15 = cuda_keccak_ROTL64(*a04, 27);
        *a04 = cuda_keccak_ROTL64(*a24, 14);
        *a24 = cuda_keccak_ROTL64(*a21, 2);
        *a21 = cuda_keccak_ROTL64(*a08, 55);
        *a08 = cuda_keccak_ROTL64(*a16, 45);
        *a16 = cuda_keccak_ROTL64(*a05, 36);
        *a05 = cuda_keccak_ROTL64(*a03, 28);
        *a03 = cuda_keccak_ROTL64(*a18, 21);
        *a18 = cuda_keccak_ROTL64(*a17, 15);
        *a17 = cuda_keccak_ROTL64(*a11, 10);
        *a11 = cuda_keccak_ROTL64(*a07, 6);
        *a07 = cuda_keccak_ROTL64(*a10, 3);
        *a10 = c1;

        /* Chi */
        c0 = *a00 ^ (~*a01 & *a02);
        c1 = *a01 ^ (~*a02 & *a03);
        *a02 ^= ~*a03 & *a04;
        *a03 ^= ~*a04 & *a00;
        *a04 ^= ~*a00 & *a01;
        *a00 = c0;
        *a01 = c1;

        c0 = *a05 ^ (~*a06 & *a07);
        c1 = *a06 ^ (~*a07 & *a08);
        *a07 ^= ~*a08 & *a09;
        *a08 ^= ~*a09 & *a05;
        *a09 ^= ~*a05 & *a06;
        *a05 = c0;
        *a06 = c1;

        c0 = *a10 ^ (~*a11 & *a12);
        c1 = *a11 ^ (~*a12 & *a13);
        *a12 ^= ~*a13 & *a14;
        *a13 ^= ~*a14 & *a10;
        *a14 ^= ~*a10 & *a11;
        *a10 = c0;
        *a11 = c1;

        c0 = *a15 ^ (~*a16 & *a17);
        c1 = *a16 ^ (~*a17 & *a18);
        *a17 ^= ~*a18 & *a19;
        *a18 ^= ~*a19 & *a15;
        *a19 ^= ~*a15 & *a16;
        *a15 = c0;
        *a16 = c1;

        c0 = *a20 ^ (~*a21 & *a22);
        c1 = *a21 ^ (~*a22 & *a23);
        *a22 ^= ~*a23 & *a24;
        *a23 ^= ~*a24 & *a20;
        *a24 ^= ~*a20 & *a21;
        *a20 = c0;
        *a21 = c1;

        /* Iota */
        *a00 ^= CUDA_KECCAK_CONSTS[i];
    }
}


__device__ __forceinline__ void cuda_keccak_absorb(cuda_keccak_ctx_t *ctx, BYTE* const in)
{

    LONG offset = 0;
	
	#pragma unroll 2
    for (LONG i = 0; i < ctx->absorb_round; ++i) {
        ctx->state[i] ^= cuda_keccak_leuint64(in + offset);
        offset += 8;
    }

    cuda_keccak_permutations(ctx);
}

__device__ __forceinline__ void cuda_keccak_pad(cuda_keccak_ctx_t *ctx)
{
    ctx->q[ctx->bits_in_queue >> 3] |= (1L << (ctx->bits_in_queue & 7));

    if (++(ctx->bits_in_queue) == ctx->rate_bits) {
        cuda_keccak_absorb(ctx, ctx->q);
        ctx->bits_in_queue = 0;
    }

    LONG full = ctx->bits_in_queue >> 6;
    LONG partial = ctx->bits_in_queue & 63;

    LONG offset = 0;
    for (int i = 0; i < full; ++i) {
        ctx->state[i] ^= cuda_keccak_leuint64(ctx->q + offset);
        offset += 8;
    }

    if (partial > 0) {
        LONG mask = (1L << partial) - 1;
        ctx->state[full] ^= cuda_keccak_leuint64(ctx->q + offset) & mask;
    }

    ctx->state[(ctx->rate_bits - 1) >> 6] ^= 9223372036854775808ULL;/* 1 << 63 */

    cuda_keccak_permutations(ctx);
    cuda_keccak_extract(ctx);

    ctx->bits_in_queue = ctx->rate_bits;
}

/*
 * Digestbitlen must be 128 224 256 288 384 512
 */
__device__ void cuda_keccak_init(cuda_keccak_ctx_t *ctx, const WORD digestbitlen)
{
    memset(ctx, 0, sizeof(cuda_keccak_ctx_t));
    ctx->sha3_flag = 0;
    ctx->digestbitlen = digestbitlen;
    ctx->rate_bits = 1600 - ((ctx->digestbitlen) << 1);
    ctx->rate_BYTEs = ctx->rate_bits >> 3;
    ctx->absorb_round = ctx->rate_bits >> 6;
    ctx->bits_in_queue = 0;
}

/*
 * Digestbitlen must be 224 256 384 512
 */
__device__ void cuda_keccak_sha3_init(cuda_keccak_ctx_t *ctx, const WORD digestbitlen)
{
    cuda_keccak_init(ctx, digestbitlen);
    ctx->sha3_flag = 1;
}

__device__ void cuda_keccak_update(cuda_keccak_ctx_t *ctx, BYTE* const in, const LONG inlen)
{
    int64_t BYTEs = ctx->bits_in_queue >> 3;
    int64_t count = 0;
	int64_t partial = 0;
    while (count < inlen) {
        if (BYTEs == 0 && count <= ((int64_t)(inlen - ctx->rate_BYTEs))) {
            do {
                cuda_keccak_absorb(ctx, in + count);
                count += ctx->rate_BYTEs;
            } while (count <= ((int64_t)(inlen - ctx->rate_BYTEs)));
        } else {
            partial = cuda_keccak_MIN(ctx->rate_BYTEs - BYTEs, inlen - count);
            memcpy(ctx->q + BYTEs, in + count, partial);

            BYTEs += partial;
            count += partial;

            if (BYTEs == ctx->rate_BYTEs) {
                cuda_keccak_absorb(ctx, ctx->q);
                BYTEs = 0;
            }
        }
    }
    ctx->bits_in_queue = BYTEs << 3;
}

__device__ void cuda_keccak_final(cuda_keccak_ctx_t *ctx, BYTE *out)
{
    if (ctx->sha3_flag) {
        int mask = (1 << 2) - 1;
        ctx->q[ctx->bits_in_queue >> 3] = (BYTE)(0x02 & mask);
        ctx->bits_in_queue += 2;
    }

    cuda_keccak_pad(ctx);
    LONG i = 0;

    while (i < ctx->digestbitlen) {
        if (ctx->bits_in_queue == 0) {
            cuda_keccak_permutations(ctx);
            cuda_keccak_extract(ctx);
            ctx->bits_in_queue = ctx->rate_bits;
        }

        LONG partial_block = cuda_keccak_UMIN(ctx->bits_in_queue, ctx->digestbitlen - i);
        memcpy(out + (i >> 3), ctx->q + (ctx->rate_BYTEs - (ctx->bits_in_queue >> 3)), partial_block >> 3);
        ctx->bits_in_queue -= partial_block;
        i += partial_block;
    }
}

__global__ void calculate(int timestamp) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    BYTE data[32] = {0};
    memcpy(data, &tid, 4);

    hiprandState state;
    hiprand_init((unsigned long long)clock() + tid, 0, 0, &state);

    for (int i = 4; i < 64; i += 4) {
        int block = (int)(hiprand_uniform_double(&state) * 1000000);
        memcpy(data+(i)/2, &block, 4);
    }

    memcpy(data+20, &timestamp, 4);

    BYTE challenge[32] = {0};
    challenge[0] = 0x72;
    challenge[1] = 0x45;
    challenge[2] = 0x54;
    challenge[3] = 0x48;

    BYTE hash[32] = {0};
        

    for (int i=0; i <N; i++) {
        memcpy(data+22, &i, 4);
        CUDA_KECCAK_CTX ctx;

        cuda_keccak_init(&ctx, 256);
        cuda_keccak_update(&ctx, data, 32);
        cuda_keccak_update(&ctx, challenge, 32);
        cuda_keccak_final(&ctx, hash);

      if (hash[0] == 0x00 && hash[1] == 0x77 && hash[2] == 0x77 && hash[3] == 0x77 && hash[4] == 0x77 && hash[5] == 0x77) {
          printf("0x");
          for (int j = 0; j < 32; j ++) {
            printf("%02x", data[j]);
          }
          printf("\n");
      }
    }

}

int main(int argc, char **argv) {
    int gpuid = 0;
    if (argc == 2) {
        gpuid = std::atoi(argv[1]);
    }
    hipSetDevice(gpuid);
    while (true) {
            time_t currentUnixTime = std::time(nullptr);
            calculate<<<24, 256>>>(static_cast<int>(currentUnixTime));
    }
    hipDeviceSynchronize();  // not important
}

